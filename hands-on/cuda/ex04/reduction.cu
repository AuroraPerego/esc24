#include "hip/hip_runtime.h"
// C++ standard headers
#include <iostream>
#include <numeric>
#include <random>
#include <vector>

// CUDA headers
#include <hip/hip_runtime.h>

// local headers
#include "cuda_check.h"

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

#define BLOCK_SiZE 1024

// Part 4 of 8: implement the kernel
__global__ void block_sum(const int* input,
                          int* per_block_results,
                          const size_t n)
{
  __shared__ int sdata[BLOCK_SiZE];

  auto gIdx = threadIdx.x + blockDim.x * blockIdx.x;
  auto lIdx = threadIdx.x;
  if (gIdx < n)
    sdata[lIdx] = input[gIdx];

  __syncthreads();

  for (int s = 1; s < blockDim.x; s *= 2) {
    auto id = 2*s*lIdx;
    if (id < blockDim.x)
      sdata[id] += sdata[id+s];
    __syncthreads();
  }

  if (lIdx == 0) atomicAdd(per_block_results, sdata[0]);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(void)
{
  std::random_device rd; // Will be used to obtain a seed for the random engine
  std::mt19937 gen(rd()); // Standard mersenne_twister_engine seeded with rd()
  std::uniform_int_distribution<> distrib(-10, 10);
  // Create array of 256ki elements
  const int num_elements = 1 << 18;
  // Generate random input on the host
  std::vector<int> h_input(num_elements);
  for (auto& elt : h_input) {
    elt = distrib(gen);
  }

  int host_result = std::accumulate(h_input.begin(), h_input.end(), 0);
  std::cerr << "Host sum: " << host_result << std::endl;

  // Part 1 of 8: choose a device and create a CUDA stream
  CUDA_CHECK(hipSetDevice(MYDEVICE));
  hipStream_t q;
  CUDA_CHECK(hipStreamCreate(&q));

  // Part 2 of 8: copy the input data to device memory
  int* d_input;
  CUDA_CHECK(hipMallocAsync(&d_input, num_elements*sizeof(int), q));
  CUDA_CHECK(hipMemcpyAsync(d_input, h_input.data(), num_elements*sizeof(int), hipMemcpyHostToDevice, q));

  // Part 3 of 8: allocate memory for the partial sums
  // How much space does it need?
  int* device_result;
  CUDA_CHECK(hipMallocAsync(&device_result, sizeof(int), q));
  CUDA_CHECK(hipMemsetAsync(device_result, 0x00, sizeof(int), q));

  // Part 5 of 8: launch one kernel to compute, per-block, a partial sum.
  // How much shared memory does it need?
  int block_size = BLOCK_SiZE;
  int num_blocks = (num_elements + block_size - 1) / block_size;
  block_sum<<<num_blocks, block_size, 0, q>>>(d_input, device_result, num_elements);
  CUDA_CHECK(hipGetLastError());

  // Part 7 of 8: copy the result back to the host
  host_result = 0;
  CUDA_CHECK(hipMemcpyAsync(&host_result, device_result, sizeof(int), hipMemcpyDeviceToHost, q));

  CUDA_CHECK(hipStreamSynchronize(q));
  std::cout << "Device sum: " << host_result << std::endl;

  // Part 8 of 8: deallocate device memory and destroy the CUDA stream
  CUDA_CHECK(hipFreeAsync(d_input, q));
  CUDA_CHECK(hipFreeAsync(device_result, q));
  CUDA_CHECK(hipStreamDestroy(q));

  return 0;
}
