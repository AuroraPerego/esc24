#include "hip/hip_runtime.h"
// C++ standard headers
#include <cassert>
#include <iostream>
#include <vector>

// CUDA headers
#include <hip/hip_runtime.h>

// local headers
#include "cuda_check.h"

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Part 2 of 4: implement the kernel
__global__ void kernel(int* a, int dimx, int dimy)
{
  auto const x = threadIdx.x + blockDim.x * blockIdx.x;
  auto const y = threadIdx.y + blockDim.y * blockIdx.y;
  if (x < dimx and y < dimy) {
    a[x+dimx*y] = x+dimx*y;
  }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main()
{
  CUDA_CHECK(hipSetDevice(MYDEVICE));

  // Create a CUDA stream to execute asynchronous operations on this device
  hipStream_t queue;
  CUDA_CHECK(hipStreamCreate(&queue));

  // Part 1 and 4 of 4: set the dimensions of the matrix
  int dimx = 19;
  int dimy = 67;

  // Allocate enough memory on the host
  std::vector<int> h_a(dimx * dimy);
  int num_bytes = dimx * dimy * sizeof(int);

  // Pointer for the device memory
  int* d_a;

  // Allocate enough memory on the device
  CUDA_CHECK(hipMallocAsync(&d_a, num_bytes, queue));

  // Part 2 of 4: define grid and block size and launch the kernel
  dim3 numberOfBlocks, numberOfThreadsPerBlock;
  numberOfThreadsPerBlock.x = 32;
  numberOfThreadsPerBlock.y = 32;
  numberOfBlocks.x  = (numberOfThreadsPerBlock.x+dimx-1)/numberOfThreadsPerBlock.x;
  numberOfBlocks.y  = (numberOfThreadsPerBlock.y+dimy-1)/numberOfThreadsPerBlock.y;

  kernel<<<numberOfBlocks, numberOfThreadsPerBlock, 0, queue>>>(d_a, dimx, dimy);
  CUDA_CHECK(hipGetLastError());

  // Device to host copy
  CUDA_CHECK(hipMemcpyAsync(h_a.data(), d_a, num_bytes, hipMemcpyDeviceToHost, queue));

  // Free the device memory
  CUDA_CHECK(hipFreeAsync(d_a, queue));


  // Wait for all asynchronous operations to complete
  CUDA_CHECK(hipStreamSynchronize(queue));


  // verify the data returned to the host is correct
  for (int row = 0; row < dimy; ++row) {
    for (int col = 0; col < dimx; ++col) {
      assert(h_a[row * dimx + col] == row * dimx + col);
    }
  }

  // Destroy the CUDA stream
  CUDA_CHECK(hipStreamDestroy(queue));

  // If the program makes it this far, then the results are correct and
  // there are no run-time errors.  Good work!
  std::cout << "Correct!" << std::endl;

  return 0;
}
